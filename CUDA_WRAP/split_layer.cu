#include "split_layer.h"

#include "../cells.h"
#include "../mesh.h"


int copyLayerToLayer(beamLayer *to,beamLayer *from)

   hipMemcpy((*to)->Ex,(*from)->Ex,sizeof(double)*Ny*Nz,hipMemcpyDeviceToDevice);
   hipMemcpy((*to)->Ey,(*from)->Ey,sizeof(double)*Ny*Nz,hipMemcpyDeviceToDevice);
   hipMemcpy((*to)->Ez,(*from)->Ez,sizeof(double)*Ny*Nz,hipMemcpyDeviceToDevice);

   hipMemcpy((*to)->Bx,(*from)->Bx,sizeof(double)*Ny*Nz,hipMemcpyDeviceToDevice);
   hipMemcpy((*to)->By,(*from)->By,sizeof(double)*Ny*Nz,hipMemcpyDeviceToDevice);
   hipMemcpy((*to)->Bz,(*from)->Bz,sizeof(double)*Ny*Nz,hipMemcpyDeviceToDevice);
   
   hipMemcpy((*to)->Jx,(*from)->Jx,sizeof(double)*Ny*Nz,hipMemcpyDeviceToDevice);
   hipMemcpy((*to)->Jy,(*from)->Jy,sizeof(double)*Ny*Nz,hipMemcpyDeviceToDevice);
   hipMemcpy((*to)->Jz,(*from)->Jz,sizeof(double)*Ny*Nz,hipMemcpyDeviceToDevice);

   hipMemcpy((*to)->particles,(*from)->particles,np*sizeof(beamParticle),hipMemcpyDeviceToDevice);
   
   return 0;
}
