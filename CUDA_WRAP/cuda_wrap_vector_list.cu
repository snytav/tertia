#include "hip/hip_runtime.h"
#include "cuda_wrap_vector_list.h"
#include "diagnostic_print.h"
#include "../run_control.h"

#include <hipblas.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
//#include <cutil.h>



       double *d_fft_of_Ex, *d_fft_of_Ey, *d_fft_of_Ez, *d_fft_of_Bx, *d_fft_of_By, *d_fft_of_Bz,
              *d_fft_of_Jx, *d_fft_of_Jy, *d_fft_of_Jz, *d_fft_of_Rho,
              *d_fft_of_JxP, *d_fft_of_JyP, *d_fft_of_JzP, *d_fft_of_RhoP,
              *d_fft_of_JxBeam, *d_fft_of_JyBeam, *d_fft_of_JzBeam, *d_fft_of_RhoBeam,
              *d_fft_of_JxBeamP, *d_fft_of_JyBeamP, *d_fft_of_JzBeamP, *d_fft_of_RhoBeamP,
              *d_fft_of_ExRho, *d_fft_of_EyRho, *d_fft_of_EzRho;
	      
double *d_fft_of_ExP, *d_fft_of_EyP, *d_fft_of_EzP, *d_fft_of_BxP, *d_fft_of_ByP, *d_fft_of_BzP;  
	      
double *d_rEx,*d_rEy,*d_rEz,*d_rBx,*d_rBy,*d_rBz,*d_rJx,*d_rJy,*d_rJz,*d_rRhoBeam,*d_rJxBeam,*d_rRho;


int compare_vector_from_device(int n,double *h_v,double *d_v,char *s)
{
   double *h_copy = (double *)malloc(n*sizeof(double)),dmax = 0.0,t;
   
   hipMemcpy(h_copy,d_v,n*sizeof(double),hipMemcpyDeviceToHost);
   
   for(int i = 0; i < n;i++)
   {
       if((t = fabs(h_v[i] - h_copy[i])) > dmax) dmax = t;
   }

   for(int i = 0; i < n;i++)
   {
       if((t = fabs(h_v[i] - h_copy[i])) > dmax*0.5) printf("%s %5d delta %15.5e device %25.15e host %25.15e \n",s,i,t, h_copy[i],h_v[i]);;
   }
   
   printf("%s i = 0 host %25.15e device %25.15e 3host,device %25.15e,%25.15e \n",s,h_v[0],h_copy[0],h_v[3],h_copy[3]);
   printf("%s %15.5e \n",s,dmax);
   free(h_copy);
   return 0;
}

//k2_dens_inv,ky,kz,  k2_inv,r,ky_k2_Jy, kz_k2_Jz,ky_k2_Jz,kz_k2_Jy, jx_ky,jx_kz

int CUDA_WRAP_device_alloc(
int a_size,
double **d_a1,
double **d_a2,
double **d_a3,
double **d_a4,
double **d_a5,
double **d_a6,
double **d_a7,
double **d_a8,
double **d_a9,
double **d_a10,
double **d_a11,
double **d_a12,
double **d_a13,
double **d_a14,
double **d_a15,
double **d_a16,
double **d_a17,
double **d_a18,
double **d_a19,
double **d_a20,
double **d_a21,
double **d_a22,
double **d_a23,
double **d_a24,
double **d_a25 
)
{
   hipMalloc((void**)d_a1,sizeof(double)*a_size);
   hipMalloc((void**)d_a2,sizeof(double)*a_size);
   hipMalloc((void**)d_a3,sizeof(double)*a_size);
   hipMalloc((void**)d_a4,sizeof(double)*a_size);

   hipMalloc((void**)d_a5,sizeof(double)*a_size);
   hipMalloc((void**)d_a6,sizeof(double)*a_size);
   hipMalloc((void**)d_a7,sizeof(double)*a_size);
   hipMalloc((void**)d_a8,sizeof(double)*a_size);

   hipMalloc((void**)d_a9,sizeof(double)*a_size);
   hipMalloc((void**)d_a10,sizeof(double)*a_size);
   hipMalloc((void**)d_a11,sizeof(double)*a_size);
   hipMalloc((void**)d_a12,sizeof(double)*a_size);

   hipMalloc((void**)d_a13,sizeof(double)*a_size);
   hipMalloc((void**)d_a14,sizeof(double)*a_size);
   hipMalloc((void**)d_a15,sizeof(double)*a_size);
   hipMalloc((void**)d_a16,sizeof(double)*a_size);
   
   hipMalloc((void**)d_a17,sizeof(double)*a_size);
   
   hipMalloc((void**)d_a18,sizeof(double)*a_size);
   hipMalloc((void**)d_a19,sizeof(double)*a_size);
   hipMalloc((void**)d_a20,sizeof(double)*a_size);
   
   hipMalloc((void**)d_a21,sizeof(double)*a_size);
   hipMalloc((void**)d_a22,sizeof(double)*a_size);
   hipMalloc((void**)d_a23,sizeof(double)*a_size);
   hipMalloc((void**)d_a24,sizeof(double)*a_size);
   hipMalloc((void**)d_a25,sizeof(double)*a_size);
   
   
   hipMemset(*d_a1, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a2, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a3, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a4, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a5, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a6, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a7, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a8, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a9, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a10, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a11, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a12, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a13, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a14, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a15, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a16, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a17, 0.0,sizeof(double)*a_size);
   
   hipMemset(*d_a18, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a19, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a20, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a21, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a22, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a23, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a24, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a25, 0.0,sizeof(double)*a_size);
   


 

//   puts("END ALLOC ================================================================");
   return 0;
}



int CUDA_WRAP_copy_all_vectors_to_device(
int a_size,
double *a1,
double *a2,
double *a3,
double *a4,
double *a5,
double *a6,
double *a7,
double *a8,
double *a9,
double *a10,
double *a11,
double *a12,
double *a13,
double *a14,
double *a15,
double *a16,
double *a17,
/*
double *a18,
double *a19,
double *a20,
double *a21,
double *a22,
double *a23,
double *a24,
double *a25,
*/
double *d_a1,
double *d_a2,
double *d_a3,
double *d_a4,
double *d_a5,
double *d_a6,
double *d_a7,
double *d_a8,
double *d_a9,
double *d_a10,
double *d_a11,
double *d_a12,
double *d_a13,
double *d_a14,
double *d_a15,
double *d_a16,
double *d_a17
/*,
double *d_a18,
double *d_a19,
double *d_a20,
double *d_a21,
double *d_a22,
double *d_a23,
double *d_a24,
double *d_a25
*/
)
{
   //puts("BEGIN COPY ======================================================================================");
   //exit(0);
   hipblasSetVector(a_size,sizeof(double),a1,1,d_a1,1); 
//   compare_vector_from_device(a_size,a1,d_a1,"copy1");
 //  puts("in devcpy0.5");
//   exit(0);

   hipblasSetVector(a_size,sizeof(double),a2,1,d_a2,1); 
   //compare_vector_from_device(a_size,a2,d_a2,"copy2");
   hipblasSetVector(a_size,sizeof(double),a3,1,d_a3,1); 
   //compare_vector_from_device(a_size,a3,d_a3,"copy3");
   hipblasSetVector(a_size,sizeof(double),a4,1,d_a4,1); 
   //compare_vector_from_device(a_size,a4,d_a4,"copy4");
   hipblasSetVector(a_size,sizeof(double),a5,1,d_a5,1); 
   //compare_vector_from_device(a_size,a5,d_a5,"copy5");

   //puts("in devcpy0.51");
   //exit(0);

   
   hipblasSetVector(a_size,sizeof(double),a6,1,d_a6,1); 
   //compare_vector_from_device(a_size,a6,d_a6,"copy6");
   hipblasSetVector(a_size,sizeof(double),a7,1,d_a7,1); 
   //compare_vector_from_device(a_size,a7,d_a7,"copy7");
   hipblasSetVector(a_size,sizeof(double),a8,1,d_a8,1); 
   //compare_vector_from_device(a_size,a8,d_a8,"copy8");
   hipblasSetVector(a_size,sizeof(double),a9,1,d_a9,1); 
   //compare_vector_from_device(a_size,a9,d_a9,"copy9");
   hipblasSetVector(a_size,sizeof(double),a10,1,d_a10,1); 
   //compare_vector_from_device(a_size,a10,d_a10,"copy10");

   //puts("in devcpy1");
  // exit(0);
   
   hipblasSetVector(a_size,sizeof(double),a11,1,d_a11,1); 
   //compare_vector_from_device(a_size,a11,d_a11,"copy11");
   //puts("in devcpy1A");
  // exit(0);
   
   hipblasSetVector(a_size,sizeof(double),a12,1,d_a12,1); 
   //compare_vector_from_device(a_size,a12,d_a12,"copy12");
   
   //puts("in devcpy1B");
//   exit(0);
   
   hipblasSetVector(a_size,sizeof(double),a13,1,d_a13,1); 
   //compare_vector_from_device(a_size,a13,d_a13,"copy13");
   
   //puts("in devcpy1C");
   //exit(0);
   
   hipblasSetVector(a_size,sizeof(double),a14,1,d_a14,1); 
   //compare_vector_from_device(a_size,a14,d_a14,"copy14");
   
   //puts("in devcpy1D");
   //exit(0);

   //hipblasSetVector(a_size,sizeof(double),a15,1,d_a15,1); 
   hipMemcpy(d_a15,a15,a_size*sizeof(double),hipMemcpyHostToDevice);
   //compare_vector_from_device(a_size,a15,d_a15,"copy15");
   
   //puts("in devcpy1E");
 //  exit(0);
   //printf("a16 %e\n",a16[0]);
//   hipblasSetVector(a_size,sizeof(double),a16,1,d_a16,1); 
   hipMemcpy(d_a16,a16,a_size*sizeof(double),hipMemcpyHostToDevice);

   hipMemcpy(d_a17,a17,a_size*sizeof(double),hipMemcpyHostToDevice);
   //compare_vector_from_device(a_size,a16,d_a16,"copy16 v before");

   //puts("END COPY ================================================================================================");
  // exit(0);
   
  /*
   hipblasSetVector(a_size,sizeof(double),a17,1,d_a17,1); 
   hipblasSetVector(a_size,sizeof(double),a18,1,d_a18,1); 
   hipblasSetVector(a_size,sizeof(double),a19,1,d_a19,1); 
   hipblasSetVector(a_size,sizeof(double),a20,1,d_a20,1); 
   
   hipblasSetVector(a_size,sizeof(double),a21,1,d_a21,1); 
   hipblasSetVector(a_size,sizeof(double),a22,1,d_a22,1); 
   hipblasSetVector(a_size,sizeof(double),a23,1,d_a23,1); 
   hipblasSetVector(a_size,sizeof(double),a24,1,d_a24,1); 
   hipblasSetVector(a_size,sizeof(double),a25,1,d_a25,1); 
   */
   return 0;
}

int CUDA_WRAP_copy_all_vectors_to_host(
int a_size,
double *a1,
double *a2,
double *a3,
double *a4,
double *a5,
double *a6,
double *a7,
double *a8,
double *a9,
double *a10,
double *a11,
double *a12,
double *a13,
double *a14,
double *a15,
double *a16,
/*
double *a17,
double *a18,
double *a19,
double *a20,
double *a21,
double *a22,
double *a23,
double *a24,
double *a25,
*/
double *d_a1,
double *d_a2,
double *d_a3,
double *d_a4,
double *d_a5,
double *d_a6,
double *d_a7,
double *d_a8,
double *d_a9,
double *d_a10,
double *d_a11,
double *d_a12,
double *d_a13,
double *d_a14,
double *d_a15,
double *d_a16
/*,
double *d_a17,
double *d_a18,
double *d_a19,
double *d_a20,
double *d_a21,
double *d_a22,
double *d_a23,
double *d_a24,
double *d_a25
*/
)
{
  
   puts("in");
   hipblasGetVector(a_size,sizeof(double),d_a1,1,a1,1); 
   hipblasGetVector(a_size,sizeof(double),d_a2,1,a2,1); 
   hipblasGetVector(a_size,sizeof(double),d_a3,1,a3,1); 
   hipblasGetVector(a_size,sizeof(double),d_a4,1,a4,1); 
   hipblasGetVector(a_size,sizeof(double),d_a5,1,a5,1); 

   hipblasGetVector(a_size,sizeof(double),d_a6,1,a6,1); 
   hipblasGetVector(a_size,sizeof(double),d_a7,1,a7,1); 
   hipblasGetVector(a_size,sizeof(double),d_a8,1,a8,1); 
   hipblasGetVector(a_size,sizeof(double),d_a9,1,a9,1); 
   hipblasGetVector(a_size,sizeof(double),d_a10,1,a10,1); 

   hipblasGetVector(a_size,sizeof(double),d_a11,1,a11,1); 
   hipblasGetVector(a_size,sizeof(double),d_a12,1,a12,1); 
   hipblasGetVector(a_size,sizeof(double),d_a13,1,a13,1); 
   hipblasGetVector(a_size,sizeof(double),d_a14,1,a14,1); 
   hipblasGetVector(a_size,sizeof(double),d_a15,1,a15,1); 
   
   hipblasGetVector(a_size,sizeof(double),d_a16,1,a16,1); 
   
/*   
   hipblasGetVector(a_size,sizeof(double),d_a17,1,a17,1); 
   hipblasGetVector(a_size,sizeof(double),d_a18,1,a18,1); 
   hipblasGetVector(a_size,sizeof(double),d_a19,1,a19,1); 
   hipblasGetVector(a_size,sizeof(double),d_a20,1,a20,1); 

   hipblasGetVector(a_size,sizeof(double),d_a21,1,a21,1); 
   hipblasGetVector(a_size,sizeof(double),d_a22,1,a22,1); 
   hipblasGetVector(a_size,sizeof(double),d_a23,1,a23,1); 
   hipblasGetVector(a_size,sizeof(double),d_a24,1,a24,1); 
   hipblasGetVector(a_size,sizeof(double),d_a25,1,a25,1); 
*/

   
   return 0;
}

int CUDA_WRAP_device_free(
double *d_a1,
double *d_a2,
double *d_a3,
double *d_a4,
double *d_a5,
double *d_a6,
double *d_a7,
double *d_a8,
double *d_a9,
double *d_a10,
double *d_a11,
double *d_a12,
double *d_a13,
double *d_a14,
double *d_a15,
double *d_a16,
double *d_a17,
double *d_a18,
double *d_a19,
double *d_a20,
double *d_a21,
double *d_a22,
double *d_a23,
double *d_a24,
double *d_a25
)
{
   hipFree(d_a1);
   hipFree(d_a2);
   hipFree(d_a3);
   hipFree(d_a4);
   hipFree(d_a5);

   hipFree(d_a6);
   hipFree(d_a7);
   hipFree(d_a8);
   hipFree(d_a9);
   hipFree(d_a10);

   hipFree(d_a11);
   hipFree(d_a12);
   hipFree(d_a13);
   hipFree(d_a14);
   hipFree(d_a15);

   hipFree(d_a16);
   hipFree(d_a17);
   hipFree(d_a18);
   hipFree(d_a19);
   hipFree(d_a20);

   hipFree(d_a21);
   hipFree(d_a22);
   hipFree(d_a23);
   hipFree(d_a24);
   hipFree(d_a25);
   
   return 0;
}


void CUDA_WRAP_free(double *d)
{
    hipFree(d);
}

int CUDA_WRAP_verify_all_vectors_on_host(
int a_size,
double *a1,double *d_a1,char *s1,
double *a2,double *d_a2,char *s2,
double *a3,double *d_a3,char *s3,
double *a4,double *d_a4,char *s4,
double *a5,double *d_a5,char *s5,
double *a6,double *d_a6,char *s6,
double *a7,double *d_a7,char *s7,
double *a8,double *d_a8,char *s8,
double *a9,double *d_a9,char *s9,
double *a10,double *d_a10,char *s10,
double *a11,double *d_a11,char *s11,
double *a12,double *d_a12,char *s12,
double *a13,double *d_a13,char *s13,
double *a14,double *d_a14,char *s14,
double *a15,double *d_a15,char *s15,
double *a16,double *d_a16,char *s16,

double *a17,double *d_a17,char *s17,
double *a18,double *d_a18,char *s18,
double *a19,double *d_a19,char *s19,
double *a20,double *d_a20,char *s20,
double *a21,double *d_a21,char *s21,
double *a22,double *d_a22,char *s22

)
{
    puts("BEGIN VERIFY =========================================================================================");
    compare_vector_from_device(a_size,a15,d_a15,"in ver 15");
    
    compare_vector_from_device(a_size,a1,d_a1,s1);
    compare_vector_from_device(a_size,a2,d_a2,s2);    
    
    compare_vector_from_device(a_size,a3,d_a3,s3);
    compare_vector_from_device(a_size,a4,d_a4,s4);    
    
    compare_vector_from_device(a_size,a5,d_a5,s5);
    compare_vector_from_device(a_size,a6,d_a6,s6);    
    
    
    
    compare_vector_from_device(a_size,a7,d_a7,s7);
    compare_vector_from_device(a_size,a8,d_a8,s8);    
    
    compare_vector_from_device(a_size,a9,d_a9,s9);
    compare_vector_from_device(a_size,a10,d_a10,s10);    
    
    compare_vector_from_device(a_size,a11,d_a11,s11);
    compare_vector_from_device(a_size,a12,d_a12,s12);  
    
    
    
    compare_vector_from_device(a_size,a13,d_a13,s13);
        
    compare_vector_from_device(a_size,a14,d_a14,s14);    
    compare_vector_from_device(a_size,a15,d_a15,s15);
    compare_vector_from_device(a_size,a16,d_a16,s16);    

    compare_vector_from_device(a_size,a17,d_a17,s17);    
    compare_vector_from_device(a_size,a18,d_a18,s18);    
    compare_vector_from_device(a_size,a19,d_a19,s19);    
    compare_vector_from_device(a_size,a20,d_a20,s20);    
    compare_vector_from_device(a_size,a21,d_a21,s21);    
    compare_vector_from_device(a_size,a22,d_a22,s22);    
    
    puts("END VERIFY ========================================================================================="); 
    
    return 0;
}

int CUDA_WRAP_verify_all_vectors_on_hostReal(
int a_size,
double *a1,double *d_a1,char *s1,
double *a2,double *d_a2,char *s2,
double *a3,double *d_a3,char *s3,
double *a4,double *d_a4,char *s4,
double *a5,double *d_a5,char *s5,
double *a6,double *d_a6,char *s6,
double *a7,double *d_a7,char *s7,
double *a8,double *d_a8,char *s8,
double *a9,double *d_a9,char *s9,
double *a10,double *d_a10,char *s10,
double *a11,double *d_a11,char *s11,
double *a12,double *d_a12,char *s12,
double *a13,double *d_a13,char *s13,
double *a14,double *d_a14,char *s14,
double *a15,double *d_a15,char *s15,
double *a16,double *d_a16,char *s16,

double *a17,double *d_a17,char *s17,
double *a18,double *d_a18,char *s18,
double *a19,double *d_a19,char *s19,
double *a20,double *d_a20,char *s20,
double *a21,double *d_a21,char *s21,
double *a22,double *d_a22,char *s22

)
{
    puts("BEGIN VERIFY =========================================================================================");
    compare_vector_from_device(a_size,a15,d_a15,"in ver 15");
    
    compare_vector_from_device(a_size,a1,d_a1,s1);
    compare_vector_from_device(a_size,a2,d_a2,s2);    
    
    compare_vector_from_device(a_size,a3,d_a3,s3);
    compare_vector_from_device(a_size,a4,d_a4,s4);    
    
    compare_vector_from_device(a_size,a5,d_a5,s5);
    compare_vector_from_device(a_size,a6,d_a6,s6);    
    
    
    
    compare_vector_from_device(a_size,a7,d_a7,s7);
    compare_vector_from_device(a_size,a8,d_a8,s8);    
    
    compare_vector_from_device(a_size,a9,d_a9,s9);
    compare_vector_from_device(a_size,a10,d_a10,s10);    
    
    compare_vector_from_device(a_size,a11,d_a11,s11);
    compare_vector_from_device(a_size,a12,d_a12,s12);  
    
    
    puts("END VERIFY ========================================================================================="); 
    
    return 0;
}



int CUDA_WRAP_copy_all_real_vectors_to_device(
int a_size,
double *a1,
double *a2,
double *a3,
double *a4,
double *a5,
double *a6,
double *a7,
double *a8,
double *a9,
double *a10,
double *a11,
double *d_a1,
double *d_a2,
double *d_a3,
double *d_a4,
double *d_a5,
double *d_a6,
double *d_a7,
double *d_a8,
double *d_a9,
double *d_a10,
double *d_a11
)
{
   int err[10];
   //puts("BEGIN COPY ======================================================================================");
   //exit(0);
   err[0] = hipblasSetVector(a_size,sizeof(double),a1,1,d_a1,1); 
//   compare_vector_from_device(a_size,a1,d_a1,"copy1");
 //  puts("in devcpy0.5");
//   exit(0);

   err[1] = hipblasSetVector(a_size,sizeof(double),a2,1,d_a2,1); 
   //compare_vector_from_device(a_size,a2,d_a2,"copy2");
   err[2] = hipblasSetVector(a_size,sizeof(double),a3,1,d_a3,1); 
   //compare_vector_from_device(a_size,a3,d_a3,"copy3");
   err[3] = hipblasSetVector(a_size,sizeof(double),a4,1,d_a4,1); 
   //compare_vector_from_device(a_size,a4,d_a4,"copy4");
   err[4] = hipblasSetVector(a_size,sizeof(double),a5,1,d_a5,1); 
   //compare_vector_from_device(a_size,a5,d_a5,"copy5");

   //puts("in devcpy0.51");
   //exit(0);

   
   err[5] = hipblasSetVector(a_size,sizeof(double),a6,1,d_a6,1); 
   //compare_vector_from_device(a_size,a6,d_a6,"copy6");
   err[6] = hipblasSetVector(a_size,sizeof(double),a7,1,d_a7,1); 
   //compare_vector_from_device(a_size,a7,d_a7,"copy7");
   err[7] = hipblasSetVector(a_size,sizeof(double),a8,1,d_a8,1); 
   //compare_vector_from_device(a_size,a8,d_a8,"copy8");
   err[8] = hipblasSetVector(a_size,sizeof(double),a9,1,d_a9,1);      
   
   //err[9] = hipblasSetVector(a_size,sizeof(double),a10,1,d_a10,1); 
   err[9] = hipMemcpy(d_a10,a10,a_size*sizeof(double),hipMemcpyHostToDevice);
   hipMemcpy(d_a11,a11,a_size*sizeof(double),hipMemcpyHostToDevice);
   return 0;
}

int CUDA_WRAP_EMERGENCY_COPY(int ny,int nz,double *d_x,double *x)
{
#ifndef CUDA_WRAP_EMERGENCY_MATRIX_COPY
   return 0;
#endif
  
   hipMemcpy(d_x,x,ny*nz*sizeof(double),hipMemcpyHostToDevice);
   
   CUDA_DEBUG_printDdevice_matrix(ny,nz,d_x,"EMERGENCY ");
   
   return 0;
}

int CUDA_WRAP_device_real_alloc(
int a_size,
double **d_a1,
double **d_a2,
double **d_a3,
double **d_a4,
double **d_a5,
double **d_a6,
double **d_a7,
double **d_a8,
double **d_a9,
double **d_a10,
double **d_a11
)
{
   hipMalloc((void**)d_a1,sizeof(double)*a_size);
   hipMalloc((void**)d_a2,sizeof(double)*a_size);
   hipMalloc((void**)d_a3,sizeof(double)*a_size);
   hipMalloc((void**)d_a4,sizeof(double)*a_size);

   hipMalloc((void**)d_a5,sizeof(double)*a_size);
   hipMalloc((void**)d_a6,sizeof(double)*a_size);
   hipMalloc((void**)d_a7,sizeof(double)*a_size);
   hipMalloc((void**)d_a8,sizeof(double)*a_size);

   hipMalloc((void**)d_a9,sizeof(double)*a_size);
   hipMalloc((void**)d_a10,sizeof(double)*a_size);

   hipMalloc((void**)d_a11,sizeof(double)*a_size);
   
   hipMemset(*d_a1, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a2, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a3, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a4, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a5, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a6, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a7, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a8, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a9, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a10, 0.0,sizeof(double)*a_size);
   hipMemset(*d_a11, 0.0,sizeof(double)*a_size);

   
   return 0;
}

int CUDA_WRAP_deviceSetZero(
int a_size,
double *d_a1,
double *d_a2,
double *d_a3,
double *d_a4,
double *d_a5,
double *d_a6,
double *d_a7,
double *d_a8,
double *d_a9,
double *d_a10,
double *d_a11,
double *d_a12,
double *d_a13,
double *d_a14,
double *d_a15,
double *d_a16,
double *d_a17,
double *d_a18,
double *d_a19,
double *d_a20,
double *d_a21,
double *d_a22,
double *d_a23,
double *d_b1,
double *d_b2,
double *d_b3,
double *d_b4,
double *d_b5,
double *d_b6,
double *d_b7,
double *d_b8,
double *d_b9
/*,
double **d_b10,
double **d_b11
*/
/*
double **d_a24,
double **d_a25 */
)
{
   hipMemset(d_a1, 0.0,sizeof(double)*a_size);
   hipMemset(d_a2, 0.0,sizeof(double)*a_size);
   hipMemset(d_a3, 0.0,sizeof(double)*a_size);
   hipMemset(d_a4, 0.0,sizeof(double)*a_size);
   hipMemset(d_a5, 0.0,sizeof(double)*a_size);
   hipMemset(d_a6, 0.0,sizeof(double)*a_size);
   hipMemset(d_a7, 0.0,sizeof(double)*a_size);
   hipMemset(d_a8, 0.0,sizeof(double)*a_size);
   hipMemset(d_a9, 0.0,sizeof(double)*a_size);
   hipMemset(d_a10, 0.0,sizeof(double)*a_size);
   hipMemset(d_a11, 0.0,sizeof(double)*a_size);
   hipMemset(d_a12, 0.0,sizeof(double)*a_size);
   hipMemset(d_a13, 0.0,sizeof(double)*a_size);
   hipMemset(d_a14, 0.0,sizeof(double)*a_size);
   hipMemset(d_a15, 0.0,sizeof(double)*a_size);
   hipMemset(d_a16, 0.0,sizeof(double)*a_size);
   hipMemset(d_a17, 0.0,sizeof(double)*a_size);
   
   hipMemset(d_a18, 0.0,sizeof(double)*a_size);
   hipMemset(d_a19, 0.0,sizeof(double)*a_size);
   hipMemset(d_a20, 0.0,sizeof(double)*a_size);
   hipMemset(d_a21, 0.0,sizeof(double)*a_size);
   hipMemset(d_a22, 0.0,sizeof(double)*a_size);
   hipMemset(d_a23, 0.0,sizeof(double)*a_size);
   
   hipMemset(d_b1, 0.0,sizeof(double)*a_size);
   hipMemset(d_b2, 0.0,sizeof(double)*a_size);
   
   hipMemset(d_b3, 0.0,sizeof(double)*a_size);
   hipMemset(d_b4, 0.0,sizeof(double)*a_size);
   hipMemset(d_b5, 0.0,sizeof(double)*a_size);
   hipMemset(d_b6, 0.0,sizeof(double)*a_size);
   hipMemset(d_b7, 0.0,sizeof(double)*a_size);
   hipMemset(d_b8, 0.0,sizeof(double)*a_size);
   hipMemset(d_b9, 0.0,sizeof(double)*a_size);
/*   hipMemset(*d_b10, 0.0,sizeof(double)*a_size);
   hipMemset(*d_b11, 0.0,sizeof(double)*a_size);
  */ 

/*
   hipMalloc((void**)d_a17,sizeof(double)*a_size);
   hipMalloc((void**)d_a18,sizeof(double)*a_size);
   hipMalloc((void**)d_a19,sizeof(double)*a_size);
   hipMalloc((void**)d_a20,sizeof(double)*a_size);

   hipMalloc((void**)d_a21,sizeof(double)*a_size);
   hipMalloc((void**)d_a22,sizeof(double)*a_size);
   hipMalloc((void**)d_a23,sizeof(double)*a_size);
   hipMalloc((void**)d_a24,sizeof(double)*a_size);

   hipMalloc((void**)d_a25,sizeof(double)*a_size);
  */ 

//   puts("END ALLOC ================================================================");
   return 0;
}

